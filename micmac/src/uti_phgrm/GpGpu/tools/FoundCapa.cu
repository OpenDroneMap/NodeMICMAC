
#include <hip/hip_runtime.h>
#include <stdio.h>


int main()
{
    int NbDevice = 0;

    if (hipSuccess != hipGetDeviceCount(&NbDevice))

        return -1;


    if (!NbDevice)

        return -1;


    for (int device = 0; device < NbDevice; ++device)
    {
        hipDeviceProp_t propri;

        if (hipSuccess != hipGetDeviceProperties(&propri, device))
        {
            continue;
        }
        printf("%d.%d ", propri.major, propri.minor);
    }

    return 0;

}
