#include "hip/hip_runtime.h"
#include "GpGpu/GpGpu_CommonHeader.h"
#include "GpGpu/GpGpu_Data.h"
#include "GpGpu/helper_math_extented.cuh"

#define __GPU_CONSTANT  __constant__
#define __GPU_GLOBAL
#define __GPU_KERNEL    __global__
#define __GPU_THREADX   threadIdx.x
