#include "hip/hip_runtime.h"
#include "GpGpu_CUDA_Define.cu" // DON'T MOVE THIS LINE!!!

#define FACTOR 4

__GPU_CONSTANT  int hw[] = {1,2,5,6,9};


__GPU_KERNEL void kMultTab(__GPU_GLOBAL int * out,  int t)
{
    size_t tid = __GPU_THREADX;

	//int2 dd = make_int2(50,20);

	//out[tid] = FACTOR*t*hw[tid] + dd.x;

	out[tid] = sgpu::__div<32>(tid*64);
}


#ifdef CUDA_ENABLED
extern void kMultTab()
{
    dim3	threads( 5, 1, 1);
    dim3	blocks(1, 1, 1);

    int* buffer     = ((CData<int>*)CGpGpuContext<cudaContext>::arg(0))->pData();
    int* pFactor    = ((int*)CGpGpuContext<cudaContext>::arg(1));

    kMultTab<<<blocks, threads>>>(buffer,*pFactor);
    getLastCudaError("kMultTab kernel failed");
}
#endif
