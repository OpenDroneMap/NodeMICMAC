#include "hip/hip_runtime.h"
#ifndef _OPTIMISATION_KERNEL_Z_H_
#define _OPTIMISATION_KERNEL_Z_H_

#include "GpGpu/GpGpu_StreamData.cuh"
#include "GpGpu/SData2Optimize.h"

// On pourrait imaginer un buffer des tailles calculer en parallel
// SIZEBUFFER[threadIdx.x] = count(lI[threadIdx.x]);



__device__ void GetConeZ(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    aDz.x =   aZ_Prev.x-aZ;
    if (aZ != aZ_Next.x)
        aDz.x = max(aDz.x,-MaxDeltaZ);

    aDz.y = aZ_Prev.y-1-aZ;
    if (aZ != aZ_Next.y-1)
        aDz.y = min(aDz.y,MaxDeltaZ);

    if (aDz.x > aDz.y)
        if (aDz.y <0)
            aDz.x = aDz.y;
        else
            aDz.y = aDz.x;
}

__device__ void BasicComputeIntervaleDelta
              (
                  short2 & aDz,
                  int aZ,
                  int MaxDeltaZ,
                  short2 aZ_Prev
              )
{
   aDz.x = max(-MaxDeltaZ,aZ_Prev.x-aZ);
   aDz.y = min(MaxDeltaZ,aZ_Prev.y-1-aZ);
}

inline __device__ uint minR(uint *sMin, uint &globalMin){ // TODO attention ajout de inline
    ushort  thread2;
    uint    temp;
    //
    int nTotalThreads = WARPSIZE;	// Total number of threads, rounded up to the next power of two

    while(nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1);	// divide by two
        // only the first half of the threads will be active.

        if (threadIdx.x < halfPoint)
        {
            thread2 = threadIdx.x + halfPoint;
            // Skipping the fictious threads blockDim.x ... blockDim_2-1
            if (thread2 < blockDim.x)
            {
                // Get the shared value stored by another thread
                temp = sMin[thread2];
                if (temp < sMin[threadIdx.x])
                    sMin[threadIdx.x] = temp;
            }
        }
        // Reducing the binary tree size by two:
        nTotalThreads = halfPoint;
    }

    const uint minus = sMin[0];

    if(minus < globalMin) globalMin = minus;

    return minus;
}

template<bool sens> __device__
inline uint __choose(uint kav,uint kar)
{
	return 0;
}

template<> __device__
inline uint __choose<true>(uint kav,uint kar)
{
	return kav;
}

template<> __device__
inline uint __choose<false>(uint kav,uint kar)
{
	return kar;
}

template<bool sens> __device__
inline ushort __choose(ushort kav,ushort kar)
{
	return 0;
}

template<> __device__
inline ushort __choose<true>(ushort kav,ushort kar)
{
	return kav;
}

template<> __device__
inline ushort __choose<false>(ushort kav,ushort kar)
{
	return kar;
}

template<bool sens> __device__
inline short __choose(short kav,short kar)
{
	return 0;
}

template<> __device__
inline short __choose<true>(short kav,short kar)
{
	return kav;
}

template<> __device__
inline short __choose<false>(short kav,short kar)
{
	return kar;
}

template<bool autoMask> __device__
inline void getIntervale(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev){}

template<> __device__
inline void getIntervale<true>(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    BasicComputeIntervaleDelta(aDz,aZ,MaxDeltaZ,aZ_Prev);
}

template<> __device__
inline void getIntervale<false>(short2 & aDz, int aZ, int MaxDeltaZ, short2 aZ_Next, short2 aZ_Prev)
{
    GetConeZ(aDz,aZ,MaxDeltaZ,aZ_Next,aZ_Prev);
}

template<bool autoMask> __device__
inline uint getCostInit(uint maskCost,uint costInit,bool mask){return 0;}


template<> __device__
inline uint getCostInit<true>(uint maskCost,uint costInit,bool mask)
{
   return mask ? maskCost : costInit;
}

template<> __device__
inline uint getCostInit<false>(uint maskCost,uint costInit,bool mask)
{
   return costInit;
}

template<bool autoMask> __device__
inline void connectMask(uint &costMin,uint costInit, uint prevDefCor, ushort costTransDefMask,bool mask){}


template<> __device__
inline void connectMask<true>(uint &costMin,uint costInit, uint prevDefCor, ushort costTransDefMask,bool mask)
{
    if(!mask)
        costMin = min(costMin, costInit + prevDefCor  + costTransDefMask );
}

template<bool sens> __device__
inline short __delta()
{
	return 0;
}

template<> __device__
inline short __delta<true>()
{
	return 0;
}

template<> __device__
inline short __delta<false>()
{
	return -WARPSIZE + 1;
}


template<bool sens> __device__
inline void __autoMask(uint &prevDefCor,const ushort &cDefCor,uint &prevMinCost,uint &prevMinCostCells, const uint &globMinFCost,p_ReadLine &p,SimpleStream<uint>  &streamDefCor)
{
	//				uint defCor = prevDefCor + cDefCor;

	//                if(p.prevDefCor != 0)
	//                    defCor = min(defCor,cDefCor + prevMinCostCells + p.costTransDefMask);

	//                prevDefCor = defCor - prevMinCost;

	if(p.prevDefCor != 0)
		prevDefCor = cDefCor - prevMinCost + min(prevDefCor,prevMinCostCells + p.costTransDefMask);
	else
		prevDefCor = cDefCor - prevMinCost + prevDefCor;

	prevMinCostCells = globMinFCost;

	prevMinCost = min(globMinFCost,prevDefCor);

	p.prevDefCor = cDefCor;

	if(p.tid == 0)
	{
		const ushort idGline = p.line.id + p.seg.id;
		streamDefCor.SetOrAddValue<sens>(__choose<sens>((uint)idGline , p.line.lenght  - idGline),prevDefCor,prevDefCor-cDefCor);
	}

}

template<bool sens,bool hasMask> __device__
inline void autoMask(uint &prevDefCor,const ushort &cDefCor,uint &prevMinCost,uint &prevMinCostCells, const uint &globMinFCost,p_ReadLine &p,SimpleStream<uint>  &streamDefCor)
{
	prevMinCost = globMinFCost;
}

template<> __device__
inline void autoMask<true,true>(uint &prevDefCor,const ushort &cDefCor,uint &prevMinCost,uint &prevMinCostCells, const uint &globMinFCost,p_ReadLine &p,SimpleStream<uint>  &streamDefCor)
{
	__autoMask<true>(prevDefCor,cDefCor,prevMinCost,prevMinCostCells, globMinFCost,p,streamDefCor);
}

template<> __device__
inline void autoMask<false,true>(uint &prevDefCor,const ushort &cDefCor,uint &prevMinCost,uint &prevMinCostCells, const uint &globMinFCost,p_ReadLine &p,SimpleStream<uint>  &streamDefCor)
{
	__autoMask<false>(prevDefCor,cDefCor,prevMinCost,prevMinCostCells, globMinFCost,p,streamDefCor);
}


template<bool sens,bool hasMask> __device__
void connectCellsLine(
                SimpleStream<short3>    &streamIndex,
                SimpleStream<uint>      &streamFCost,
                SimpleStream<ushort>    &streamICost,
                SimpleStream<uint>      &streamDefCor,
                short3     *S_Bf_Index,
                ushort     *ST_Bf_ICost,
                uint       *S_FCost[2],
                p_ReadLine &p
)
{

	short3* ST_Bf_Index = S_Bf_Index + p.tid + __delta<sens>();

    __shared__ uint minCost[WARPSIZE];
    short2  ConeZ;
    uint globMinFCost;

    bool lined = p.line.id < p.line.lenght;

    const int regulZ  = (int)((float)10000.f*p.ZRegul);

    // Remarque
    // p.seg.id = 1 au premier passage, car simple copie des initcost


    //////////////////////////////////////////////////
    /// TODO!!!! : quel doit etre prevDefCor p.costTransDefMask + p.costDefMask ou p.costDefMask
    /////////////////////////////////////////////////
	uint         prevDefCor	=/* p.costTransDefMask + */p.prevDefCor; // TODO Voir la valeur � mettre!!!
	const ushort idGline	= p.line.id + p.seg.id;

	streamDefCor.SetOrAddValue<sens>(__choose<sens>((uint)idGline, p.line.lenght  - idGline),prevDefCor);

    uint         prevMinCostCells    = 0; // TODO cette valeur doit etre determiner
    uint         prevMinCost         = 0;

    while(lined)
    {
        while(p.seg.id < p.seg.lenght)
        {
            const short3 dTer       = S_Bf_Index[sgn(p.seg.id)];
            const short2 indexZ     = make_short2(dTer.x,dTer.y);
            const ushort cDefCor    = dTer.z;
            const bool   maskTer    = cDefCor == 0;
            const ushort dZ         = count(indexZ); // creer buffer de count
            ushort       z          = 0;
            globMinFCost            = max_cost;

            while( z < dZ)
            {                
                // Lecture du stream si le buffer est vide | TODO VERIFIER si > ou >=
                if(p.ID_Bf_Icost >= p.sizeBuffer)
                {
                    streamICost.read<sens>(ST_Bf_ICost);    //  Lecture des couts correlations
                    streamFCost.incre<sens>();              //  Pointage sur la sortie
                    p.ID_Bf_Icost = 0;                      //  Pointage la premi�re valeur du buffer des couts correlations
                }

                uint    fCostMin        = max_cost;

                uint    costInit        = getCostInit<hasMask>(500000,ST_Bf_ICost[sgn(p.ID_Bf_Icost)],maskTer);

                const ushort tZ         = z + p.stid<sens>();
				const short  Z          = __choose<sens>((short)(tZ + indexZ.x),(short)(indexZ.y - tZ - 1));
				const short  pitPrZ     = __choose<sens>((short)(Z - p.prev_Dz.x ), (short)(p.prev_Dz.y - Z - 1));

                getIntervale<hasMask>(ConeZ,Z,p.pente,indexZ,p.prev_Dz);

				const uint* prevFCost	= S_FCost[p.Id_Buf] + sgn(pitPrZ);

                ConeZ.y = min(p.sizeBuffer - pitPrZ,ConeZ.y );

                for (short i = ConeZ.x; i <= ConeZ.y; ++i) //--> TO DO cette etape n'est pas necessaire si nous sommes en dehors du masque Ter
                    fCostMin = min(fCostMin, costInit + prevFCost[i] + abs((int)i)*regulZ);

                connectMask<hasMask>(fCostMin,costInit,prevDefCor,p.costTransDefMask,maskTer);

                if(tZ < dZ && p.ID_Bf_Icost +  p.stid<sens>() < p.sizeBuffer && tZ < p.sizeBuffer)
                {                    

                    fCostMin                    -= prevMinCost;
                    minCost[p.tid]               = fCostMin;
                    S_FCost[!p.Id_Buf][sgn(tZ)]  = fCostMin;

                    streamFCost.SetOrAddValue<sens>(sgn(p.ID_Bf_Icost),fCostMin,fCostMin - costInit);                    
                }
                else
                    minCost[p.tid] = max_cost;

                minR(minCost,globMinFCost); // TODO verifier cette fonction elle peut lancer trop de fois..... Attentioncd ,inline en attendant

                const ushort pIdCost = p.ID_Bf_Icost;
                p.ID_Bf_Icost       += min(dZ - z               , WARPSIZE);
                z                   += min(p.sizeBuffer-pIdCost , WARPSIZE);

            }

			autoMask<sens,hasMask>(prevDefCor,cDefCor,prevMinCost,prevMinCostCells, globMinFCost,p,streamDefCor);

            p.prev_Dz = indexZ;
            p.seg.id++;
            p.swBuf();

        }

        p.line.id += p.seg.lenght;

        lined = p.line.id < p.line.lenght;

        if(lined)
        {
            streamIndex.read<sens>(ST_Bf_Index);
            p.seg.lenght  = min(p.line.LOver(),WARPSIZE);
            p.seg.id      = 0; // position dans le segment du stream index des Z
        }
    }
}

// TODO Passer les parametres en variable constante !!!!!!!!!!!

template<class T,bool hasMask> __global__
void Kernel_OptimisationOneDirection(ushort* g_ICost, short3* g_Index, uint* g_FCost, uint* g_DefCor, uint3* g_RecStrParam, ushort penteMax, float zReg,float zRegQuad, ushort costDefMask,ushort costTransDefMask,ushort sizeBuffer,bool hasMaskauto)
{

    extern __shared__ float sharedMemory[];

    ushort*   S_BuffICost0 = (ushort*)  sharedMemory;
    uint*     S_BuffFCost0 = (uint*)    &S_BuffICost0[sizeBuffer + 2*WARPSIZE];
    uint*     S_BuffFCost1 = (uint*)    &S_BuffFCost0[sizeBuffer + 2*WARPSIZE];
    short3*   S_BuffIndex  = (short3*)  &S_BuffFCost1[sizeBuffer + 2*WARPSIZE];
    uint*     pit_Id       = (uint*)    &S_BuffIndex[WARPSIZE];
    uint*     pit_Stream   = pit_Id + 1;

    p_ReadLine p(threadIdx.x,penteMax,zReg,zRegQuad,costDefMask,costTransDefMask,sizeBuffer,hasMaskauto);

    uint*    S_BuffFCost[2] = {S_BuffFCost0 + WARPSIZE,S_BuffFCost1 + WARPSIZE};
    ushort*  S_BuffICost    = S_BuffICost0 + WARPSIZE + p.tid;

    if(!threadIdx.x)
    {
        *pit_Stream          = g_RecStrParam[blockIdx.x].x;
        *pit_Id              = g_RecStrParam[blockIdx.x].y;
    }

    __syncthreads();

    p.line.lenght   = g_RecStrParam[blockIdx.x].z;
    p.seg.lenght    = min(p.line.LOver(),WARPSIZE);

    SimpleStream<ushort>    streamICost(    g_ICost     + *pit_Stream   ,sizeBuffer);
    SimpleStream<uint>      streamFCost(    g_FCost     + *pit_Stream   ,sizeBuffer);
    SimpleStream<short3>    streamIndex(    g_Index     + *pit_Id       ,WARPSIZE);
    SimpleStream<uint>      streamDefCor(   g_DefCor    + *pit_Id       ,WARPSIZE);

	if(p.tid == 0)
		streamDefCor.SetValue(0,0); // car la premiere ligne n'est calculer
	// Attention voir pour le retour arriere

	streamICost.read<eAVANT>(S_BuffICost);
	streamIndex.read<eAVANT>(S_BuffIndex + p.tid);

    p.prev_Dz       = make_short2(S_BuffIndex[0].x,S_BuffIndex[0].y);
    p.prevDefCor    = S_BuffIndex[0].z;
    p.ID_Bf_Icost   = count(p.prev_Dz);

    for (ushort i = 0; i < p.ID_Bf_Icost - p.tid; i+=WARPSIZE)
    {
        S_BuffFCost[p.Id_Buf][i + p.tid] = S_BuffICost[i];
        streamFCost.SetValue(i,S_BuffICost[i]);
    }

	connectCellsLine<eAVANT,hasMask>(streamIndex,streamFCost,streamICost,streamDefCor,S_BuffIndex,S_BuffICost,S_BuffFCost,p);

    streamIndex.ReverseIncre<eARRIERE>();
    streamFCost.incre<eAVANT>();
    streamFCost.reverse<eARRIERE>();

    S_BuffFCost[0]  += sizeBuffer;
    S_BuffFCost[1]  += sizeBuffer;
    S_BuffICost     += sizeBuffer - WARPSIZE;

    streamICost.readFrom<eARRIERE>(S_BuffFCost[p.Id_Buf] + p.tid, sizeBuffer - p.ID_Bf_Icost);
    streamICost.ReverseIncre<eARRIERE>();

    p.reverse(S_BuffIndex,sizeBuffer);

    if(p.ID_Bf_Icost > sizeBuffer)
    {
        p.ID_Bf_Icost -= sizeBuffer;
        streamICost.read<eARRIERE>(S_BuffICost);
        streamFCost.incre<eARRIERE>();
    }

    uint* locFCost = S_BuffFCost[p.Id_Buf] - p.stid<eARRIERE>();

    for (ushort i = 0; i < sizeBuffer; i+=WARPSIZE)
        locFCost[-i] = S_BuffICost[-i];

	connectCellsLine<eARRIERE,hasMask>( streamIndex,streamFCost,streamICost,streamDefCor,S_BuffIndex + WARPSIZE - 1,S_BuffICost,S_BuffFCost,p);
}

extern "C" void Gpu_OptimisationOneDirection(Data2Optimiz<CuDeviceData3D> &d2O)
{
    ushort  deltaMax         = d2O.penteMax();
    float   zReg             = (float)d2O.zReg();
    float   zRegQuad         = d2O.zRegQuad();
    ushort  costDefMask      = d2O.CostDefMasked();
    ushort  costTransDefMask = d2O.CostTransMaskNoMask();
    bool    hasMaskauto      = d2O.hasMaskAuto();

    dim3 Threads(WARPSIZE,1,1);
    dim3 Blocks(d2O.NBlines(),1,1);

    ushort sizeBuff = min(d2O.DzMax(),4096);  //NAPPEMAX;
    ushort cacheLin = sizeBuff + 2 * WARPSIZE;

    // Calcul de l'allocation dynamique de la memoire partag�e
    uint   sizeSharedMemory =
            cacheLin * sizeof(ushort)   + // S_BuffICost0
            cacheLin * sizeof(uint)     + // S_BuffFCost0
            cacheLin * sizeof(uint)     + // S_BuffFCost1
            WARPSIZE * sizeof(short3)   + // S_BuffIndex
          //  WARPSIZE * sizeof(uint)     + // S_BuffDefCor
            sizeof(uint)                + // pit_Id
            sizeof(uint);                 // pit_Stream


	if(hasMaskauto)
		Kernel_OptimisationOneDirection< uint,true ><<<Blocks,Threads,sizeSharedMemory>>>
																						(
																							d2O.pInitCost(),
																							d2O.pIndex(),
																							d2O.pForceCostVol(),
																							d2O.pDefCor(),
																							d2O.pParam(),
																							deltaMax,
																							zReg,
																							zRegQuad,
																							costDefMask,
																							costTransDefMask,
																							sizeBuff,
																							hasMaskauto
																							);
	else
		Kernel_OptimisationOneDirection< uint,false ><<<Blocks,Threads,sizeSharedMemory>>>
																						 (
																							 d2O.pInitCost(),
																							 d2O.pIndex(),
																							 d2O.pForceCostVol(),
																							 d2O.pDefCor(),
																							 d2O.pParam(),
																							 deltaMax,
																							 zReg,
																							 zRegQuad,
																							 costDefMask,
																							 costTransDefMask,
																							 sizeBuff,
																							 hasMaskauto
																							 );


    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
    {        
        printf("Error CUDA Gpu_OptimisationOneDirection\n");
        printf("%s",hipGetErrorString(err));
        DUMP(d2O.NBlines());
        DUMP(sizeSharedMemory);
        DUMP(d2O.DzMax());
    }

    getLastCudaError("TestkernelOptiOneDirection failed");

}


#endif //_OPTIMISATION_KERNEL_Z_H_

